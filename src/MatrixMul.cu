#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
//#include <hip/hip_runtime.h>
#include <stdlib.h>
#include "Rinternals.h"    

#define BLOCK_SIZE 32

// treat it as C code
extern "C" {
    SEXP matrixMultiply(SEXP ma,SEXP mb);
}
//#define BLOCKSIZE 192
__global__ void 
matrixMulCUDA(int *A,int *B,int *C,int m,int n,int k)
{
    int Cvalue=0;
    int row=blockIdx.y*blockDim.y+threadIdx.y;
    int col=blockIdx.x*blockDim.x+threadIdx.x;
    if(col<k && row<m){
    for(int e=0;e<n;++e)
    {
       Cvalue+=A[row*n+e]*B[e*k+col];
    }
    C[row*k+col]=Cvalue;
    }
}

SEXP matrixMultiply(SEXP ma,SEXP mb)
{
    SEXP maDim=getAttrib(ma,R_DimSymbol);
    SEXP mbDim=getAttrib(mb,R_DimSymbol);
    int Ax=INTEGER(maDim)[0];
    int Ay=INTEGER(maDim)[1];
    int Bx=INTEGER(mbDim)[0];
    int By=INTEGER(mbDim)[1];
    if(Ay!=Bx)
    {
    printf("error:matrix A's colsize not equal to B's rowsize.");
    exit(0);
    }
    int m=Ax;
    int n=Ay;
    int k=By;
    printf("Ax:%d\n",Ax);
    printf("Ay:%d\n",Ay);
    printf("Bx:%d\n",Bx);
    printf("By:%d\n",By);
    //double* tempA=REAL(ma);
    //double* tempB=REAL(mb);
    //float h_A[Ax*Ay];
    //float h_B[Bx*By];
    //for(int i=0;i<Ax*Ay;i++)
    //   h_A[i]=(float)tempA[i];
    //for(int j=0;j<Bx*By;j++)
    //   h_B[j]=(float)tempB[j];
    //float* h_C=(float *) malloc(Bx*Ay);
    //printf("%f",h_A[0]);
    int *h_A=INTEGER(ma);
    int *h_B=INTEGER(mb);
    int *h_C=(int *) malloc(m*k*sizeof(int));
    //printf("%d",h_A[1]);
    int *d_A;
    int *d_B;
    int *d_C;
    int size_A=m*n*sizeof(int);
    int size_B=n*k*sizeof(int);
    int size_C=m*k*sizeof(int);
    //printf("size_A:%d",size_A);
    //printf("size_B:%d",size_B);
    hipMalloc((void **)&d_A,size_A);
    hipMalloc((void **)&d_B,size_B);
    hipMalloc((void **)&d_C,size_C);

    hipMemcpy(d_A,h_A,size_A,hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_B,size_B,hipMemcpyHostToDevice);
    
    
    //dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
    //dim3 dimGrid(By/dimBlock.x,Ax/dimBlock.y);
    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
    dim3 dimGrid((k+BLOCK_SIZE-1)/BLOCK_SIZE,(m+BLOCK_SIZE-1)/BLOCK_SIZE);
    matrixMulCUDA<<<dimGrid,dimBlock>>>(d_A,d_B,d_C,m,n,k);
    hipDeviceSynchronize();
    hipMemcpy(h_C,d_C,size_C,hipMemcpyDeviceToHost);
    //hipDeviceSynchronize();
    //printf("%d",h_C[1]);
    SEXP reVec=PROTECT(allocMatrix(REALSXP,Ax,By));
    for(int i=0;i<Ax;i++)
        for(int j=0;j<By;j++)
        REAL(reVec)[i+Ax*j]=h_C[i+Ax*j];
    UNPROTECT(1);
    return reVec;
    //return ScalarReal(222);
}

